#include "Static/KTruss/KTruss.cuh"
#include "KTrussOperators.cuh"
#include "KTrussSupport.cu"

#include <iostream>

using namespace std;

namespace hornets_nest {

void kTrussOneIteration(HornetGraph& hornet,
                        const triangle_t*  __restrict__ output_triangles,
                        int threads_per_block,
                        int number_blocks,
                        int shifter,
                        int thread_blocks,
                        int blockdim,
                        HostDeviceVar<KTrussData>& hd_data);

//==============================================================================

KTruss::KTruss(HornetGraph& hornet) : StaticAlgorithm(hornet) {
    hd_data().active_queue.initialize(hornet);
    originalNE = hornet.nE();
    originalNV = hornet.nV();

}

KTruss::~KTruss() {
    release();
}

void KTruss::setInitParameters(int tsp, int nbl, int shifter,
                               int blocks, int sps) {
    hd_data().tsp     = tsp;
    hd_data().nbl     = nbl;
    hd_data().shifter = shifter;
    hd_data().blocks  = blocks;
    hd_data().sps     = sps;
}

void KTruss::init(){
    gpu::allocate(hd_data().is_active,            originalNV);
    gpu::allocate(hd_data().offset_array,         originalNV + 1);
    gpu::allocate(hd_data().triangles_per_vertex, originalNV);
    gpu::allocate(hd_data().triangles_per_edge,   originalNE);
    gpu::allocate(hd_data().src,                  originalNE);
    gpu::allocate(hd_data().dst,                  originalNE);
    gpu::allocate(hd_data().counter,              1);
    gpu::allocate(hd_data().active_vertices,      1);
    reset();
}

void KTruss::copyOffsetArrayHost(const vert_t* host_offset_array) {
    // host::copyToDevice(host_offset_array, hornet.nV() + 1,
    //                    hd_data().offset_array);
    hipMemcpy(hd_data().offset_array,host_offset_array,(originalNV + 1)*sizeof(vert_t), hipMemcpyHostToDevice);

}

void KTruss::copyOffsetArrayDevice(vert_t* device_offset_array){
    // host::copyToDevice(device_offset_array, hornet.nV() + 1,
    //                    hd_data().offset_array);
    hipMemcpy(hd_data().offset_array,device_offset_array,(originalNV + 1)*sizeof(vert_t), hipMemcpyDeviceToDevice);
}

vert_t KTruss::getMaxK() {
    return hd_data().max_K;
}

//==============================================================================

void KTruss::reset() {
    hipMemset(hd_data().counter,0, sizeof(int));//hd_data().counter = 0
    hd_data().num_edges_remaining      = originalNE;
    hd_data().full_triangle_iterations = 0;

    resetEdgeArray();
    resetVertexArray();
}

void KTruss::resetVertexArray() {
    gpu::memsetZero(hd_data().triangles_per_vertex, originalNV);
}

void KTruss::resetEdgeArray() {

    // gpu::memsetZero(hd_data().triangles_per_edge, hornet.nE());
    gpu::memsetZero(hd_data().triangles_per_edge, originalNE);
}

void KTruss::release() {
    gpu::free(hd_data().is_active);
    gpu::free(hd_data().offset_array);
    gpu::free(hd_data().triangles_per_edge);
    gpu::free(hd_data().triangles_per_vertex);
    gpu::free(hd_data().counter);
    gpu::free(hd_data().active_vertices);

    hd_data().is_active            = nullptr;
    hd_data().offset_array         = nullptr;
    hd_data().triangles_per_edge   = nullptr;
    hd_data().triangles_per_vertex = nullptr;
    hd_data().counter              = nullptr;
    hd_data().active_vertices      = nullptr;

}

//==============================================================================

void KTruss::run() {
    hd_data().max_K = 3;
    int  iterations = 0;

    while (true) {
        // if(hd_data().max_K >= 1000)
        //     break;
        //std::cout << hd_data().num_edges_remaining << std::endl;
        bool need_stop = false;

        printf("Number of remaining edges %ld\n",hd_data().num_edges_remaining);

        bool      more = findTrussOfK(need_stop);

        //if (more == false && need_stop) {
        // if(hornet.nE()==0){
        //     hd_data().max_K--;
        //     break;
        // }

        if (hd_data().num_edges_remaining <= 0) {
            hd_data().max_K--;
            break;
        }
        hd_data().max_K++;

        iterations++;
    }
    std::cout << "iterations " << iterations << std::endl;
    cout << "Found the maximal KTruss at : " << hd_data().max_K << endl;
    std::cout << "The number of full triangle counting iterations is  : "
             << hd_data().full_triangle_iterations << std::endl;
}

void KTruss::runForK(int max_K) {
    hd_data().max_K = max_K;

    bool exit_on_first_iteration;
    findTrussOfK(exit_on_first_iteration);
}

bool KTruss::findTrussOfK(bool& stop) {
    forAllVertices(hornet, Init { hd_data });
    resetEdgeArray();
    resetVertexArray();
 
    hipMemset(hd_data().counter,0, sizeof(int));//hd_data().counter = 0

    int h_active_vertices = originalNV;
    // hipMemcpy(hd_data().active_vertices,&h_active_vertices, sizeof(int),hipMemcpyHostToDevice);

    // int sum_deleted_edges = 0;
    stop = true;

    while (h_active_vertices > 0) {


        std::cout << "MaxK =  " << hd_data().max_K << std::endl;

        hd_data().full_triangle_iterations++;

        kTrussOneIteration(hornet, hd_data().triangles_per_vertex,
                           hd_data().tsp, hd_data().nbl,
                           hd_data().shifter,
                           hd_data().blocks, hd_data().sps,
                           hd_data);
        CHECK_ERROR("Crashing after tricount")

        forAllVertices(hornet, FindUnderK { hd_data });
        CHECK_ERROR("Crashing after findk")

        int h_counter;
        hipMemcpy(&h_counter,hd_data().counter, sizeof(int),hipMemcpyDeviceToHost);

        std::cout << "Current number of deleted edges is " << h_counter << std::endl;

        if (h_counter != 0) {
              UpdatePtr ptr(h_counter, hd_data().src, hd_data().dst);
              Update batch_update(ptr);
              hornet.erase(batch_update);

            // BatchUpdate batch_update(hd_data().src, hd_data().dst,
            //                          hd_data().counter, gpu::BatchType::DEVICE);

            // hornet.deleteEdgeBatch(batch_update);
        }
        else{
            return false;
        }

        if(hd_data().num_edges_remaining==0)
            printf("BAAAA\n");

        hd_data().num_edges_remaining -= h_counter;
        if(hd_data().num_edges_remaining==0)
            printf("GAAAA\n");


        // Resetting the number of active vertices before check
        hipMemset(hd_data().active_vertices,0, sizeof(int));
        forAllVertices(hornet, CountActive { hd_data });
        // Getting the number of active vertices

        hipMemcpy(&h_active_vertices, hd_data().active_vertices,sizeof(int),hipMemcpyDeviceToHost);
        printf("Number of active vertices %d\n",h_active_vertices);

        // CHECK_ERROR("Crashing after counting")

        // if (h_counter == 0) 
        //     return false;



        // hd_data.sync();
        resetEdgeArray();
        resetVertexArray();

        hipMemset(hd_data().counter,0, sizeof(int));//hd_data().counter = 0
        stop = false;




    }
    return true;
}

//==============================================================================
//==============================================================================
//==============================================================================

// void KTruss::runDynamic(){
//     hd_data().max_K = 3;
//     forAllVertices(hornet, Init { hd_data });

//     resetEdgeArray();
//     resetVertexArray();

//     kTrussOneIteration(hornet, hd_data().triangles_per_vertex,
//                            hd_data().tsp, hd_data().nbl,
//                            hd_data().shifter,
//                            hd_data().blocks, hd_data().sps,
//                            hd_data);   //sub
//     hd_data.sync();
//     forAllVertices(hornet, ResetWeights { hd_data });

//     int iterations = 0;
//     while (true) {
//         //if(hd_data().max_K >= 5)
//         //    break;
//         //std::cout << "New iteration" << std::endl;
//         bool need_stop = false;
//         bool     more = findTrussOfKDynamic(need_stop);
//         CHECK_CUDA_ERROR
//         //std::cout << hd_data().num_edges_remaining << std::endl;
//         //if (more == false && need_stop) {
//         if (hd_data().num_edges_remaining <= 0) {
//             hd_data().max_K--;
//             break;
//         }
//         hd_data().max_K++;
//         iterations++;
//     }
//     //std::cout << "iterations " << iterations << std::endl;
// }

// bool KTruss::findTrussOfKDynamic(bool& stop) {
//     hd_data().counter = 0;
//     hd_data().active_queue.clear();  //queue

//     forAllVertices(hornet, QueueActive { hd_data }); //queue
//     forAllVertices(hornet, CountActive { hd_data });
//     hd_data.sync();
//     hd_data().active_queue.swap();//queue

//     stop = true;
//     while (hd_data().active_vertices > 0) {
//         forAllVertices(hornet, hd_data().active_queue,
//                        FindUnderKDynamic { hd_data });    //queue
//         hd_data.sync();
//         //std::cout << "Current number of deleted edges is "
//         //<< hd_data().counter << std::endl;

//         /*if (hd_data().counter == hd_data().num_edges_remaining) {
//             stop = true;
//             return false;
//         }*/
//         if (hd_data().counter != 0) {
//             //directly on the device
//             //auto src_array = new vert_t[hd_data().counter];
//             //auto dst_array = new vert_t[hd_data().counter];
//             //cuMemcpyToHost(hd_data().src, hd_data().counter,
//             //               src_array);
//             //cuMemcpyToHost(hd_data().dst, hd_data().counter,
//             //               dst_array);
//             //hornet::BatchInit batch_init(src_array, dst_array,
//             //                                hd_data().counter);
//             BatchUpdate batch_update(hd_data().src, hd_data().dst,
//                                      hd_data().counter, gpu::BatchType::DEVICE);
//             //batch_update.sendToDevice(batch_init);//directly on the device
//             hornet.deleteEdgeBatch(batch_update);

//             callDeviceDifferenceTriangles(hornet, batch_update,
//                                           hd_data().triangles_per_vertex,
//                                           hd_data().tsp,
//                                           hd_data().nbl,
//                                           hd_data().shifter,
//                                           hd_data().blocks,
//                                           hd_data().sps, true);
//         }
//         else
//             return false;

//         hd_data().num_edges_remaining -= hd_data().counter;
//         hd_data().active_vertices = 0;
//         hd_data().counter         = 0;

//         //allVinA_TraverseVertices<ktruss_operators::countActive>
//         //    (hornet, deviceKTrussData, hd_data().active_queue.getQueue(),
//         //     activeThisIteration);

//         forAllVertices(hornet, hd_data().active_queue, CountActive { hd_data });

//         hd_data.sync();
//         stop = false;
//     }
//     return true;
// }

// void KTruss::runForKDynamic(int max_K) {
//     hd_data().max_K = max_K;

//     forAllVertices(hornet, Init { hd_data });

//     resetEdgeArray();
//     resetVertexArray();

//     kTrussOneIteration(hornet, hd_data().triangles_per_vertex, 4,
//                        hd_data().sps / 4, 2, hd_data().blocks,
//                        hd_data().sps, hd_data);
//     hd_data.sync();

//     forAllVertices(hornet, ResetWeights { hd_data });

//     bool need_stop = false;
//     bool      more = findTrussOfKDynamic(need_stop);
// }

} // hornet_alg namespace
