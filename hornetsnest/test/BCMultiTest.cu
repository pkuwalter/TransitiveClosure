/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BetweennessCentrality/bc.cuh"
#include "Static/BetweennessCentrality/exact_bc.cuh"
#include "Static/BetweennessCentrality/approximate_bc.cuh"
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <omp.h>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>


using namespace std;
using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;



#include <hipcub/hipcub.hpp> 


#include <thrust/random.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>


#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/random/uniform_real_distribution.h>

//fill vector w/ random values;
//(specialization for floating point types)
//
template<typename T, typename IndexT = int64_t>
// std::enable_if<std::is_floating_point<T>::value>
void
random_fill(thrust::device_vector<T>& d_v, IndexT seed, T lb, T ub)
{
  thrust::counting_iterator<T> index_sequence_begin(seed);

  thrust::transform(thrust::device, index_sequence_begin, index_sequence_begin + d_v.size(),
                    d_v.begin(),
                    [lb, ub] __host__ __device__(T index) { //okay
                      // thrust::minstd_rand rng(index);
                       thrust::default_random_engine rng;
                      thrust::uniform_real_distribution<T> dist(lb, ub);
                      // thrust::uniform_real_distribution<T> dist(lb, ub);
                      rng.discard(index);
                      // return double(index+1) * dist(rng);
                      return 10000000.0/(double(index+1));
                    });

  //debug:
  //std::cout<<"double specialization...\n";
}

int floatTest() {

    const int N = 1<<25;
    int64_t seed = 0;
    thrust::device_vector<double> V1(N);
    thrust::device_vector<double> V2(N);
    thrust::device_vector<double> V3(N);

    random_fill(V1,seed,0.0,double(1));
    random_fill(V2,seed,0.0,double(1));

    // thrust::transform(thrust::device,V1.begin(), V1.end(), V2.begin(), V3.begin(), thrust::multiplies<double>());                
    thrust::transform(thrust::device,V1.begin(), V1.end(), V2.begin(), V3.begin(), thrust::plus<double>());                

    double sumS = thrust::reduce(thrust::device,  V3.begin(), V3.end(),0.0);

    cout << setprecision (25) << "double value - unsorted : " << sumS << endl;

    thrust::sort(thrust::device,V3.begin(), V3.end());

    sumS = thrust::reduce(thrust::device,  V3.begin(), V3.end(),0.0);

    cout << setprecision (25) << "double value -   sorted : " << sumS << endl;


    return 0;
}

int main(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    // for(int k=0; k<5;k++){
    //     floatTest();
    // }
    // return 1;

    // GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);
    Timer<DEVICE> TM;

    int numHardwareGPUs=8;
    int numGPUs=8;

    int numRoots = 100;
    // int numRoots = graph.nV();

    if (argc >2)
        numRoots = atoi(argv[2]);
 
    if (argc >3)
        numGPUs = atoi(argv[3]);

    hipSetDevice(0);

    bc_t *mgpuGlobalBC,*temp;
    gpu::allocate(mgpuGlobalBC, graph.nV());
    gpu::allocate(temp, graph.nV());

    hipMemset(mgpuGlobalBC,0, sizeof(bc_t)*graph.nV());

    omp_set_num_threads(numGPUs);

    bc_t* bcArray[numGPUs];

    paths_t *singleGPUSigma,*multiGPUSigma; 
    bc_t *singleGPUDelta,*multiGPUDelta; 

    gpu::allocate(singleGPUSigma, graph.nV());
    gpu::allocate(multiGPUSigma, graph.nV());
    gpu::allocate(singleGPUDelta, graph.nV());
    gpu::allocate(multiGPUDelta, graph.nV());

    TM.start();

    // Create a single Hornet Graph for each GPU
    #pragma omp parallel
    {
        int64_t thread_id = omp_get_thread_num();

        hipSetDevice(thread_id%numHardwareGPUs);

        HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                               graph.csr_out_edges());
        HornetGraph hornet_graph(hornet_init);

        vid_t* roots = new vid_t[graph.nV()/numGPUs+1];

        int i=0;
        // for(int v=thread_id; v<graph.nV(); v+=numGPUs){
        for(int v=thread_id; v<numRoots; v+=numGPUs){
            roots[i++]=v;
        }


        ApproximateBC abc(hornet_graph,roots,i);
        abc.reset();
        delete[] roots;

        abc.run();

        if(thread_id==0){
            hipMemcpy(multiGPUSigma,abc.getSigmas(),sizeof(paths_t)*graph.nV(), hipMemcpyDeviceToDevice);
            hipMemcpy(multiGPUDelta,abc.getDeltas(),sizeof(bc_t)*graph.nV(), hipMemcpyDeviceToDevice);
        }

        bcArray[thread_id] = abc.getBCScores();

        #pragma omp barrier

        #pragma omp master
        {
            hipSetDevice(0);

            for(int t=0; t<numGPUs;t++){
                hipMemcpy(temp,bcArray[t],sizeof(bc_t)*graph.nV(), hipMemcpyDeviceToDevice);
                thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), temp, mgpuGlobalBC,
                   thrust::plus<bc_t>());                
                // thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), bcArray[t], mgpuGlobalBC,
                //    thrust::plus<bc_t>());

            }
            printf("Reduction completed\n"); fflush(stdout);

        }
        // bc_t sumM = thrust::reduce(thrust::device, abc.getBCScores(),abc.getBCScores()+graph.nV(),0.0);

        #pragma omp barrier

        // cout << "Total BC scores (multi ) : " << sumM << endl;

    }
    TM.stop();
    TM.print("MultiGPU Time");

    hipSetDevice(0);

    gpu::free(temp);

    bc_t sumM = thrust::reduce(thrust::device, mgpuGlobalBC,mgpuGlobalBC+graph.nV(),0.0);

    cout << "Total BC scores (multi )   : " << sumM << endl;



    bc_t *sgpuGlobalBC,*diff;

    gpu::allocate(sgpuGlobalBC, graph.nV());
    hipMemset(sgpuGlobalBC,0, sizeof(bc_t)*graph.nV());
    gpu::allocate(diff, graph.nV());
    hipMemset(diff,0, sizeof(bc_t)*graph.nV());


    TM.start();

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());
    HornetGraph hornet_graph(hornet_init);

    vid_t* roots = new vid_t[graph.nV()];

    int i=0;
    for(int v=0; v<numRoots; v++){
        roots[i++]=v;
    }


    ApproximateBC abc(hornet_graph,roots,i);
    abc.reset();
    delete[] roots;


    abc.run();

    TM.stop();
    TM.print("SingleGPU Time");

    hipMemcpy(singleGPUSigma,abc.getSigmas(),sizeof(paths_t)*graph.nV(), hipMemcpyDeviceToDevice);
    hipMemcpy(singleGPUDelta,abc.getDeltas(),sizeof(bc_t)*graph.nV(), hipMemcpyDeviceToDevice);


    thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), abc.getBCScores(), diff,
               thrust::minus<bc_t>());                

    bc_t diffS = thrust::reduce(thrust::device,  diff, diff+graph.nV(),0.0);

    bc_t sumS = thrust::reduce(thrust::device,  abc.getBCScores(), abc.getBCScores()+graph.nV(),0.0);

    cout << "Total BC scores (single) : " << sumS << endl;

    cout << "Total difference in sum is : " << diffS << endl;




    paths_t *sigmaDiff;
    bc_t *deltaDiff;

    gpu::allocate(sigmaDiff, graph.nV());
    gpu::allocate(deltaDiff, graph.nV());

    thrust::transform(thrust::device,multiGPUSigma, multiGPUSigma+graph.nV(), singleGPUSigma, sigmaDiff, thrust::minus<bc_t>());                
    thrust::transform(thrust::device,sigmaDiff, sigmaDiff+graph.nV(), sigmaDiff, sigmaDiff, thrust::multiplies<bc_t>());                
    paths_t sumSquarePath = thrust::reduce(thrust::device,  sigmaDiff, sigmaDiff+graph.nV(),0);

    thrust::transform(thrust::device,multiGPUDelta, multiGPUDelta+graph.nV(), singleGPUDelta, deltaDiff, thrust::minus<bc_t>());                
    // thrust::transform(thrust::device,deltaDiff, deltaDiff+graph.nV(), deltaDiff, deltaDiff, thrust::multiplies<bc_t>());                
    bc_t sumSquareDelta = thrust::reduce(thrust::device,  deltaDiff, deltaDiff+graph.nV(),0.0);


    bc_t* cubsum, h_cubsum;
    gpu::allocate(cubsum, 1);

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, deltaDiff, cubsum, graph.nV());
    hipMalloc(&d_temp_storage,temp_storage_bytes);

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, deltaDiff, cubsum, graph.nV());
    hipFree(d_temp_storage);

    hipMemcpy(&h_cubsum,cubsum,sizeof(bc_t), hipMemcpyDeviceToHost);



    gpu::free(cubsum);

    cout << "Total sum square diff of sigma : " << sumSquarePath << endl;
    cout << "Total THRUSTsum square diff of delta : " << sumSquareDelta << endl;
    cout << "Total CUBsum square diff of delta : " << h_cubsum << endl;



    gpu::free(sigmaDiff);
    gpu::free(deltaDiff);



    gpu::free(multiGPUSigma);
    gpu::free(singleGPUSigma);

    gpu::free(diff);
    gpu::free(sgpuGlobalBC);
    gpu::free(mgpuGlobalBC);



}
