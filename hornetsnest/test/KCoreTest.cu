
/**
 * @brief KCore decomposition test program
 * @file
 */

#include "Static/KCore/KCore.cuh"
#include <Device/Util/Timer.cuh>
#include <Graph/GraphStd.hpp>

using namespace timer;
using namespace hornets_nest;

int main(int argc, char **argv) {
    // hipSetDevice(1);
    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vert_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
    KCoreDynamic kcore(hornet_graph);
    kcore.run();

}
