/**
 * @brief Connected-Component test program
 * @file
 */


#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <Hornet.hpp>
#include <HornetAlg.cuh>
#include <BasicTypes.hpp>

#include <Device/Util/Timer.cuh>


namespace hornets_nest{
    using string_t = char;
    using soffset_t = int;

    using HornetInit  = ::hornet::HornetInit<string_t,EMPTY,EMPTY,soffset_t>;
    // using HornetDynamicGraph = ::hornet::gpu::Hornet<string_t>;
    using HornetStaticGraph = ::hornet::gpu::HornetStatic<string_t>;

}

#define CHECK_ERROR(str) \
    {hipError_t err; err = hipGetLastError(); if(err!=0) {printf("ERROR %s:  %d %s\n", str, err, hipGetErrorString(err)); fflush(stdout); exit(0);}}

#include "offsetKernels.cuh"

template <typename HornetGraph>
int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    soffset_t wordCount   = 3;
    soffset_t letterCount = 11; 

    string_t letters[letterCount]       = {'m','y','f','i','r','s','t','t','e','s','t'};
    soffset_t wordOffsets[wordCount+1]  = {0,2,7,11};

    // HornetInit hornet_init(wordCount,letterCount,wordOffsets,letters);
    // HornetStaticGraph hgraph(hornet_init);

    int* dCounter = NULL;
    // string_t myChar = 't';
    hipMalloc((void**)&dCounter, sizeof(int));
    hipMemset(dCounter,0, sizeof(int));

    // load_balancing::BinarySearch load_balancing(hgraph);

    // forAllEdges(hgraph, hornets_nest::findAndCount {myChar, dCounter },load_balancing);



    // forAll(, hornets_nest::findAndCount {myChar, dCounter },load_balancing);

    hipEventRecord(start); 
    hipEventSynchronize(start); 
    if(argc >=2 ){
        FILE *f = fopen(argv[1], "r");
        fseek(f, 0, SEEK_END);
        long fsize = ftell(f);
        fseek(f, 0, SEEK_SET);  /* same as rewind(f); */

        string_t *h_fileInfo = (string_t *)malloc(fsize + 1);
        size_t readSize = fread(h_fileInfo, 1, fsize, f);
        fclose(f);

        h_fileInfo[fsize] = 0;    

        string_t* d_fileInfo;
        hipMalloc((string_t**)&d_fileInfo, sizeof(string_t)*fsize);
        hipMemcpy(d_fileInfo,h_fileInfo, sizeof(string_t)*fsize, hipMemcpyHostToDevice);
        hipMemset(dCounter,0, sizeof(int));

        CHECK_ERROR("Failing before allocation")
        string_t myChar='\n';
        // forAll(fsize, hornets_nest::findAndCountArray {myChar, dCounter, d_fileInfo });

        int hlineCounts = 0;
        hipMemcpy(&hlineCounts,dCounter,sizeof(int), hipMemcpyDeviceToHost);

        printf("Line Count : %d \n",hlineCounts);

        free(h_fileInfo);
        hipFree(d_fileInfo);

    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);  
    printf("%f,", milliseconds/1000.0);             



    hipFree(dCounter);


    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
    hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.

    //ret = exec<hornets_nest::HornetDynamicGraph, hornets_nest::BfsTopDown2Dynamic>(argc, argv);
    ret = exec<hornets_nest::HornetStaticGraph>(argc, argv);

    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    hornets_nest::gpu::finalizeRMMPoolAllocation();

    return ret;
}

